#include "hip/hip_runtime.h"
__global__ void cu3phsr(double *****fc3rd, double *mass, double **mesh, int iq1, double **eigVal, hipDoubleComplex ***eigVec)
{
    // fc3rd  (natom_in_uc, n_neigh_atom, 3, 3, 3)
    // eigVal (nq, nmode)
    // eigVec (nq, nmode, nmode)


    size_t iq2 = blockIdx.x;

    size_t imod1 = threadIdx.x;
    size_t imod2 = threadIdx.y;
    size_t imod3 = threadIdx.z;

}

